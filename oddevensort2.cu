#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>

#define swap(A,B) {int temp=A;A=B;B=temp;}

// Performing odd phase of odd-even sort
__global__ void oddPhaseSort(int *arr, int n) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n - 1; i += gridDim.x * blockDim.x) {
        if (i % 2 != 0) {
            if (arr[i] > arr[i + 1]) {
                swap(arr[i], arr[i + 1]);
            }
        }
    }
}

// Performing even phase of odd-even sort
__global__ void evenPhaseSort(int *arr, int n) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n - 1; i += gridDim.x * blockDim.x) {
        if (i % 2 == 0) {
            if (arr[i] > arr[i + 1]) {
                swap(arr[i], arr[i + 1]);
            }
        }
    }
}

// Init array with random values
void init_arr(int *arr, int size) {
    for (int i = 0; i < size; i++)
        arr[i] = rand() % (100 - 1);
}

// Check if array is sorted
void
sorted(int *v, int size)
{
    int a = 1, d = 1, i = 0;

    while((a == 1 || d == 1) && i < size - 1) {
        if (v[i] < v[i+1])
            d = 0;
        else if (v[i] > v[i+1])
            a = 0;
        i++;
    }

    if (a == 1)
        printf("The array is sorted in ascending order.\n");
    else if (d == 1)
        printf("The array is sorted in descending order.\n");
    else
        printf("The array is not sorted.\n");
}

void
print_arr(int *v, int size)
{
    int i;
    for (i = 0; i < size; i++) {
        printf("%d, ", v[i]);
    }
}


int main() {
    int arraySize = 1 << 19; // Define array size
    int *array, *d_array;
    size_t size = arraySize * sizeof(int);

    // Allocate memory for host and device arrays
    array = (int*)malloc(size);

    // Allocate memory on the device
    hipMalloc((void **)&d_array, arraySize * sizeof(int));

    init_arr(array, arraySize);
    // print_arr(array, arraySize);
    sorted(array, arraySize);

    // Copy input array from host to device
    hipMemcpy(d_array, array, arraySize * sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel with thread and block dimensions
    int T = 256;
    // int B = 10;
    int B = ceil((float) arraySize / T);

    clock_t t; 
    t = clock(); 

    for (int phase = 0; phase < arraySize; phase++) {
        if (phase % 2 == 0) {
            evenPhaseSort<<<B, T>>>(d_array, arraySize);
        } else {
            oddPhaseSort<<<B, T>>>(d_array, arraySize);
        }
    }

    
    // Copy the result back from the device to the host
    hipMemcpy(array, d_array, arraySize * sizeof(int), hipMemcpyDeviceToHost);
    
    t = clock() - t; 
    double time_taken = ((double)t)/CLOCKS_PER_SEC; // in seconds 
    printf("sorting took %f seconds to execute \n", time_taken); 
    // Free memory on the device
    hipFree(d_array);

    // print_arr(array, arraySize);
    sorted(array, arraySize);

    return 0;
}
