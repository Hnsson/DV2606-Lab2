/***************************************************************************
 *
 * Sequential version of Gauss-Jordan row reduction
 *
 ***************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define MAX_SIZE 4096

typedef double matrix[MAX_SIZE][MAX_SIZE];

int	N;		        /* matrix size		*/
int	maxnum;		    /* max number of element*/
int	PRINT;		    /* print switch		*/
matrix	A;		        /* matrix A		*/
double	b[MAX_SIZE];	/* vector b             */
double	y[MAX_SIZE];	/* vector y             */
double temp_d_a[MAX_SIZE*MAX_SIZE];

__global__ void gauss_jordan_p1(double *d_A, double *d_b, double *d_y, int N, int k) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i != 0) return;

    for (int j = k + 1; j < N; j++) {
        d_A[k*N + j] = d_A[k*N + j] / d_A[k*N + k];
    }
    d_y[k] = d_b[k] / d_A[k*N + k];
    d_A[k*N + k] = 1.0;
}
__global__ void gauss_jordan_p2(double *d_A, double *d_b, double *d_y, int N, int k) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i > k && i < N) {
        for(int j = k + 1; j < N; j++) {
            d_A[i*N + j] = d_A[i*N + j] - d_A[i*N + k] * d_A[k*N + j];
        }
        d_b[i] = d_b[i] - d_A[i*N + k] * d_y[k];
        d_A[i*N + k] = 0.0;
    }
}

__global__ void gauss_jordan_p3(double *d_A, double *d_b, double *d_y, int N, int k) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < k && i < N) {
        for(int j = k + 1; j < N; j++) {
            d_A[i*N + j] = d_A[i*N + j] - d_A[i*N + k] * d_A[k*N + j];
        }
        d_y[i] = d_y[i] - d_A[i*N + k] * d_y[k];
        d_A[i*N + k] = 0.0;
    }
}


/* forward declarations */
void print_matrix(void);
void init_default(void);
void print_options(void);
void read_options(int argc, char**argv);
void init_matrix(void);
void sequential_algorithm(void);

int
main(int argc, char** argv)
{
    printf("Gauss Jordan\n");

    /* INIT SETTINGS */
    init_default();
    read_options(argc, argv);
    init_matrix();
    print_options();
    // print_matrix();

    // Perform calculations using CUDA
    double *d_A;
    double *d_b;
    double *d_y;

    // Allocate memory on the GPU
    hipMalloc((void**)&d_A, sizeof(double) * MAX_SIZE * MAX_SIZE);
    hipMalloc((void**)&d_b, sizeof(double) * MAX_SIZE);
    hipMalloc((void**)&d_y, sizeof(double) * MAX_SIZE);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            temp_d_a[i*N + j] = A[i][j];
        }
    }

    // Copy data from host to device
    hipMemcpy(d_A, temp_d_a, sizeof(double) * MAX_SIZE * MAX_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(double) * MAX_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sizeof(double) * MAX_SIZE, hipMemcpyHostToDevice);

    // Define block and grid dimensions
    int threads = 256;
    int threadsPerBlock = (N * threads - 1) / threads;

    // Call the CUDA kernel function
    for (int k = 0; k < N; k++) {
        gauss_jordan_p1<<<threads, 1>>>(d_A, d_b, d_y, N, k);
        gauss_jordan_p2<<<threads, threadsPerBlock>>>(d_A, d_b, d_y, N, k);
        gauss_jordan_p3<<<threads, threadsPerBlock>>>(d_A, d_b, d_y, N, k);
    }
    hipDeviceSynchronize();

    // Copy the result back to the host
    hipMemcpy(temp_d_a, d_A, sizeof(double) * MAX_SIZE * MAX_SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, sizeof(double) * MAX_SIZE, hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, sizeof(double) * MAX_SIZE, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[i][j] = temp_d_a[i*N + j];
        }
    }

    // Free memory on the GPU
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_y);

    // Print the modified matrix and vectors
    // print_matrix();

    return 0;
}

void
print_matrix(void)
{
    int i, j;

    printf("Matrix A:\n");
    for (i = 0; i < N; i++) {
        printf("[");
        for (j = 0; j < N; j++)
            printf(" %5.2f,", A[i][j]);
        printf("]\n");
    }
    printf("Vector y:\n[");
    for (j = 0; j < N; j++)
        printf(" %5.2f,", y[j]);
    printf("]\n");
    printf("\n\n");
}

void
init_default(void)
{
    N = 2048;
    maxnum = 15.0;
    PRINT = 0;
}

void
print_options(void) {
    printf("\nsize      = %dx%d ", N, N);
    printf("\nmaxnum    = %d \n", maxnum);
    printf("printing  = %s \n\n", (PRINT == 1) ? "true" : "false");

}

void
read_options(int argc, char** argv)
{
    char* prog;

    prog = *argv;
    while (++argv, --argc > 0)
        if (**argv == '-')
            switch (*++ * argv) {
            case 'n':
                --argc;
                N = atoi(*++argv);
                break;
            case 'h':
                printf("\nHELP: try sor -u \n\n");
                exit(0);
                break;
            case 'u':
                printf("\nUsage: gaussian [-n problemsize]\n");
                printf("           [-D] show default values \n");
                printf("           [-h] help \n");
                printf("           [-m maxnum] max random no \n");
                printf("           [-P print_switch] 0/1 \n");
                exit(0);
                break;
            case 'D':
                printf("\nDefault:  n         = %d ", N);
                printf("\n          maxnum    = 5 ");
                printf("\n          P         = 0 \n\n");
                exit(0);
                break;
            case 'm':
                --argc;
                maxnum = atoi(*++argv);
                break;
            case 'P':
                --argc;
                PRINT = atoi(*++argv);
                break;
            default:
                printf("%s: ignored option: -%s\n", prog, *argv);
                printf("HELP: try %s -u \n\n", prog);
                break;
            }
}

void
init_matrix(void)
{
    int i, j;

    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            if (i == j) /* diagonal dominance */
                A[i][j] = (double)(rand() % maxnum) + 5.0;
            else
                A[i][j] = (double)(rand() % maxnum) + 1.0;
        }
    }

    /* Initialize vectors b and y */
    for (i = 0; i < N; i++) {
        b[i] = 2.0;
        y[i] = 1.0;
    }
}
